
#include <hip/hip_runtime.h>
__global__ void evaluateSymbolRegression(float* resultScore, float* result, float* programArray, float* evaluateBuffer, int* stackCountArray, int* programLength, int *maxProgramLengthFromMain, int *targetFunction, float* targetValueArray){
  // allocate buffer for processing
  const unsigned int maxProgramLength = maxProgramLengthFromMain[0];
  // extern __shared__ float copyBuffer[];

  const int index = threadIdx.x;

  int posCounter = programLength[index] - 1;

  // Copy a program array(individual) into buffer for processing
  while(posCounter >= 0){
    evaluateBuffer[index*maxProgramLength + posCounter] = 
      programArray[index*maxProgramLength + posCounter];
    posCounter--;
  }
  
  unsigned int currentProgramLength = programLength[index];
  while(currentProgramLength > 1){
    
    unsigned int currentProgramNode = 0;
    for(int i=currentProgramLength-1; i>=0; i--){
      // Remainder with 1.0 equals 0.0 (means integer),
      // stack count is less than zero,
      // detected as function node
      if(fmod((double)evaluateBuffer[index*maxProgramLength + i], 1.0) == 0.0 && 
	 (int)evaluateBuffer[index*maxProgramLength + i] > 65535 &&
	 stackCountArray[(int)evaluateBuffer[index*maxProgramLength + i] - 65535] <= 0){
	// Search Last Function Node
	currentProgramNode = i;
	break;
      }
    }

    int currentStackCount = 
      stackCountArray[(int)evaluateBuffer[index*maxProgramLength + currentProgramNode] -65535];
    unsigned int subTreeRange = 1;

    while(currentStackCount < 1){
      currentStackCount += 1;
      subTreeRange++;
    }

    float resultFromFunctionNode = 0.0;
    unsigned int functionLabel = 
      (int)evaluateBuffer[index*maxProgramLength + currentProgramNode] - 65535;

    switch(functionLabel){
    case 0: //add
      resultFromFunctionNode = 
	evaluateBuffer[index*maxProgramLength + currentProgramNode +1] 
	+ 
	evaluateBuffer[index*maxProgramLength + currentProgramNode +2];
      break;
    case 1: //sub
      resultFromFunctionNode = 
	evaluateBuffer[index*maxProgramLength + currentProgramNode +1] 
	- 
	evaluateBuffer[index*maxProgramLength + currentProgramNode +2];
      break;
    case 2: //mul
      resultFromFunctionNode = 
	evaluateBuffer[index*maxProgramLength + currentProgramNode +1] 
	*
	evaluateBuffer[index*maxProgramLength + currentProgramNode +2];
      break;
    case 3: //div
      if(evaluateBuffer[index*maxProgramLength + currentProgramNode + 1] != 0.0 &&
	 evaluateBuffer[index*maxProgramLength + currentProgramNode + 2] != 0.0){
	resultFromFunctionNode = 
	  evaluateBuffer[index*maxProgramLength + currentProgramNode +1] 
	  /
	  evaluateBuffer[index*maxProgramLength + currentProgramNode +2];
      }
      else
	resultFromFunctionNode = 1.0;
      break;
    case 4: //IfLessThenElse
      if(evaluateBuffer[index*maxProgramLength + currentProgramNode +1] > 
	 evaluateBuffer[index*maxProgramLength + currentProgramNode +2])
	resultFromFunctionNode = 
	  evaluateBuffer[index*maxProgramLength + currentProgramNode + 3];
      else
	resultFromFunctionNode =
	  evaluateBuffer[index*maxProgramLength + currentProgramNode + 4];
      break;
    case 5: //cos
      resultFromFunctionNode = 
	cos(evaluateBuffer[index*maxProgramLength + currentProgramNode + 1]);
      break;
    case 6: //sin
      resultFromFunctionNode = 
	sin(evaluateBuffer[index*maxProgramLength + currentProgramNode + 1]);
      break;
    default:
      resultFromFunctionNode = 0.0;
      break;
    }
    
    // Replace current function node with result of calc subtree
    evaluateBuffer[index*maxProgramLength + currentProgramNode] = 
      resultFromFunctionNode;
    // Copy node array after current subtree.
    for(int i=0; i<subTreeRange; i++)
      evaluateBuffer[index*maxProgramLength + currentProgramNode + 1 + i] = 
	evaluateBuffer[index*maxProgramLength + currentProgramNode + subTreeRange + i];
    // new currentProgramLength
    currentProgramLength = currentProgramLength - subTreeRange + 1;
    // Fill zero after currentProgramLength
    for(int i=index*maxProgramLength + currentProgramLength; 
	i<(index+1)*maxProgramLength;
	i++)
      evaluateBuffer[i] = 0.0;
  }
  result[index] = (float)evaluateBuffer[index*maxProgramLength] - targetValueArray[index] * evaluateBuffer[index*maxProgramLength] - targetValueArray[index];
  
  __syncthreads(); // Waiting Finish All Threads
  
 resultScore[0] = 0.0;

  if(index == 0){ // If thread ID == 0 then calc score from RSS array
      float sum = 0;
      for(unsigned int j=0; j<blockDim.x; j++){
          sum += result[j];
      }
      if(targetFunction[0] == 0){ //MSE: Mean Square Error
	resultScore[0] = (sum / blockDim.x);
      }
      else if(targetFunction[0] == 1){ //MDL: Minimum Description Length
	resultScore[0] = (sum / blockDim.x) + (programLength[0] * logf(blockDim.x));
      }
      else{
	resultScore[0] = 0.0;
      }
  }
}
